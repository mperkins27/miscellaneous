#include "hip/hip_runtime.h"
/*
 *
 * Max Perkins
 * CSCI 7332 - Parallel Algorithms Design & Analysis
 * Fall 2018
 *
 * Final Project: IDW-based Spatiotemporal Interpolation using CUDA
 *
 *
*/


#include "hip/hip_runtime.h"
#include ""
#include "helper_functions.h"

#include <iostream>
#include <string>
#include <fstream>
#include <sstream>
#include <vector>
#include <random>
#include <algorithm>

using namespace std;

typedef struct site {
	double x, y;
	double pm25;
	double d;
	int t;
} SITE;

typedef struct errs {
	double err;
	double p;
	double est_pm25;
} ERRS;

struct {
	bool operator()(SITE a, SITE b) const
	{
		return a.d < b.d;
	}
} customComp;

SITE *pm25data = NULL;
SITE *modData = NULL;

SITE *trainingData = NULL;
SITE *testData = NULL;

const int dataSize = 146125;
const int testSize = (int)dataSize / 10;
const int trainingSize = dataSize - testSize;

const double c = 0.1086;

void seqDistanceCalc(SITE *data, int dataSize, double x, double y, int t, double c);
double idwCalc(int N, double p, SITE *);
void loadData();

__global__
void distanceCalc(SITE *data, const int size, double x, double y, int t, const double c) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	double dt, dy, dx;

	if (i < size) {
		dx = data[i].x - x;
		dy = data[i].y - y;
		dt = data[i].t - t;
					
		data[i].d = (dx*dx) + (dy*dy) + (dt*dt)*(c*c);

		// 1.4^2 = 1.96, 7^2 = 49
		if (data[i].d > 1.96 || dt > 49)
			data[i].d = INFINITY;

	}
}

int main() {	
	cout << "Allocating host memory...\n";

	pm25data = (SITE *)malloc(sizeof(SITE) * dataSize);
	modData = (SITE *)malloc(sizeof(SITE) * dataSize);

	trainingData = (SITE *)malloc(sizeof(SITE) * trainingSize);
	testData = (SITE *)malloc(sizeof(SITE) * testSize);

	SITE *d_pm25data = NULL;
	SITE *d_trainingData = NULL;	
	
	hipError_t cudaStatus;

	StopWatchInterface *gpuTimer = NULL;
	sdkCreateTimer(&gpuTimer);

	StopWatchInterface *hTimer = NULL;
	sdkCreateTimer(&hTimer);	

	cout << "Reading PM 2.5 2009 data file and building array...\n";
	loadData();	

	cout << "Setting the default device and allocating device memory...\n";
	hipSetDevice(0);
	hipMalloc(&d_pm25data, dataSize * sizeof(SITE));
	hipMalloc(&d_trainingData, trainingSize * sizeof(SITE));

	cout << "Copying PM 2.5 data to device memory...\n";
	hipMemcpy(d_pm25data, pm25data, dataSize * sizeof(SITE), hipMemcpyHostToDevice);

	double x, y;	
	int t, nn, i;		

	nn = 3;

	cout << "\n";
	while(true) {
		cout << "\nc = " << to_string(c) << ", N = " << to_string(nn) << "\n";
		cout << "Enter t, y, x (or 1's to run error calculations, 0's to quit): ";		
		cin >> t >> y >> x;

		if (!x && !y && !t)
			break;
		else if (x==1 && y==1 && t==1) {
			// Run error checker						
			random_device                  rand_dev;
			mt19937                        generator(rand_dev());
			uniform_int_distribution<int>  distr(0, dataSize - 1);

			int randNums[testSize];
			
			cout << "Initializing test data...\n";
			// Get random indices in [0, dataSize)
			for (int i = 0; i < testSize; i++)
				randNums[i] = distr(generator);
			
			// Initialize test data
			for (int i = 0; i < testSize; i++)
				testData[i] = pm25data[randNums[i]];
			
			cout << "Initializing \"training\" data...\n";
			// Initialize "training" data
			int tr = 0;
			for (int i = 0; i < dataSize; i++) {
				bool found = false;
				for (int j = 0; j < testSize && !found; j++)
					found = (i == randNums[j]) ? true : false;

				if (!found && tr < trainingSize)
					trainingData[tr++] = pm25data[i];									
			}

			int n = 10;

			ERRS bestErr[10];

			double bestE = 10, worstE = 90;
			double bestP = 6, worstP = 0.5;

			cout << "Running " << to_string(n) << " calculations...\n";

			sdkResetTimer(&gpuTimer);
			sdkResetTimer(&hTimer);
			
			double hVal = 0;
			double gVal = 0;
			
			for (int i = 0; i < n; i++) {							
				cout << "Data point: " << to_string(i+1) << " out of " << to_string(n) << ": "
					 << to_string(testData[i].t) << " " << to_string(testData[i].y) << " "
					 << to_string(testData[i].x) << " " << to_string(testData[i].pm25) << " "<< endl;

				cout << "\tCopying \"training\" data to device memory...\n";
				hipMemcpy(d_trainingData, trainingData, trainingSize * sizeof(SITE), hipMemcpyHostToDevice);

				sdkStartTimer(&gpuTimer);
				cout << "\tCalculating square of the distances in parallel...\n";
				distanceCalc << < 1169, 125 >> > (d_trainingData, trainingSize, testData[i].x, testData[i].y, testData[i].t, c);
				sdkStopTimer(&gpuTimer);
				
				gVal += sdkGetTimerValue(&gpuTimer) / 1000;
				
				sdkStartTimer(&hTimer);
				cout << "\tCalculating square of the distances in serial...\n";
				seqDistanceCalc(trainingData, trainingSize, testData[i].x, testData[i].y, testData[i].t, c);
				sdkStopTimer(&hTimer);

				hVal += sdkGetTimerValue(&hTimer) / 1000;

				cout << "\tCopying results to host memory...\n";
				hipMemcpy(trainingData, d_trainingData, trainingSize * sizeof(SITE), hipMemcpyDeviceToHost);
				
				cout << "\tSorting the distances...\n";				
				sort(trainingData, trainingData + trainingSize - 1, customComp);

				double est_pm25 = 0;

				bestErr[i].err = 5;
				bestErr[i].p = 0;
				
				cout << "\tCalculating errors with 0.5 <= p <= 6, calculating best p and error...\n";
				for (double p = 0.5; p <= 6; p += 0.5) {
					est_pm25 = idwCalc(nn, p, trainingData);

					// calc error
					double err = abs(((est_pm25 - testData[i].pm25) / testData[i].pm25) * 100);

					if (err > worstE) {
						worstE = err;
						worstP = p;
					}
						
					if (err < bestE) {
						bestE = err;
						bestP = p;
					}						

					if (err < bestErr[i].err) {
						bestErr[i].err = err;
						bestErr[i].p = p;
						bestErr[i].est_pm25 = est_pm25;
					}
						
					cout << "\t\tp = " << to_string(p) << " Est: " << to_string(est_pm25) << " ";
					cout << "O: " << to_string(testData[i].pm25) << " Error: " << to_string(err) << endl;
				}		
			
				cout << "\tNearest neighbors (" << to_string(nn) << ") :\n";

				for (int i = 0; i < nn; i++) {
					cout << "\t\t";
					cout << to_string(trainingData[i].t) << " " << to_string(trainingData[i].y) << " "
						<< to_string(trainingData[i].x) << " " << to_string(trainingData[i].pm25) << " "
						<< to_string(trainingData[i].d) << "\n";
				}
			}
			

			cout << "Complete!\n ";
		
			cout << "Parallel Time: " << to_string(gVal) << " seconds\n";
			cout << "Serial Time  : " << to_string(hVal) << " seconds\n";
			cout << "Speedup      : " << to_string(hVal / gVal) << "\n";

			cout << "Best Error : " << to_string(bestE)  << " Best P: " << to_string(bestP) << endl;
			cout << "Worst Error: " << to_string(worstE) << " Worst P: " << to_string(worstP) << endl;
			
			double allP[] = { 0.5, 1.0, 1.5, 2.0, 2.5, 3.0, 3.5, 4.0, 4.5, 5.0, 5.5, 6.0};
			int freq[] = { 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0 };

			for (int j = 0; j < n; j++) {
				int index = ((double)bestErr[j].p / 0.5) - 1;

				freq[index]++;
			}

			cout << "Frequency of Best P's:\n";
			for (int i = 0; i < 12; i++) {
				if(freq[i] > 0)
					cout << to_string(allP[i]) << " - " << to_string(freq[i]) << "\n";
			}
		}
		else {			
			cout << "Calculating square of the distance in parallel...\n";

			// start the timers
			sdkResetTimer(&gpuTimer);
			sdkStartTimer(&gpuTimer);

			//distanceCalc<<< 2048, 128 >>> (d_pm25data, dataSize, x, y, t, c);
			distanceCalc <<< 1169, 125 >>> (d_pm25data, dataSize, x, y, t, c);

			sdkStopTimer(&gpuTimer);

			// get milliseconds
			double gTimerValue = sdkGetTimerValue(&gpuTimer);

			cout << "Parallel time: " << to_string(gTimerValue) << " milliseconds\n";

			// Check for any errors launching the kernel
			cudaStatus = hipGetLastError();
			if (cudaStatus != hipSuccess)
				cout << "distanceCalc kernel launch failed: "
				<< hipGetErrorString(cudaStatus) << "\n";

			cout << "Synchronizing device threads...\n";

			// hipDeviceSynchronize waits for the kernel to finish, and returns
			// any errors encountered during the launch.
			cudaStatus = hipDeviceSynchronize();
			if (cudaStatus != hipSuccess)
				cout << "hipDeviceSynchronize returned error code "
				<< to_string(cudaStatus)
				<< "after launching distanceCalc kernel!\n";

			cout << "Copying results to host memory...\n";
			hipMemcpy(modData, d_pm25data, dataSize * sizeof(SITE), hipMemcpyDeviceToHost);

			cout << "Calculating the distances sequentially...\n";
			// start the timers
			sdkResetTimer(&hTimer);
			sdkStartTimer(&hTimer);

			seqDistanceCalc(pm25data, dataSize, x, y, t, c);

			sdkStopTimer(&hTimer);

			// get milliseconds
			double sTimerValue = sdkGetTimerValue(&hTimer);

			cout << "Serial time: " << to_string(sTimerValue) << " milliseconds\n";

			cout << "Speedup is: " << to_string(sTimerValue / gTimerValue) << "\n";

			cout << "Sorting the distances...\n";
			sort(modData, modData + dataSize - 1, customComp);
			//quickSort(modData, 0, dataSize - 1);

			cout << "Nearest neighbors (" << to_string(nn) << ") :\n";

			for (i = 0; i < nn; i++) {
				cout << to_string(modData[i].t) << " " << to_string(modData[i].y) << " "
					<< to_string(modData[i].x) << " " << to_string(modData[i].pm25) << " "
					<< to_string(modData[i].d) << "\n";
			}

			double p, est_pm25;

			for (p = 0.5; p < 6.5; p += 0.5) {
				est_pm25 = idwCalc(nn, p, modData);
				cout << "p = " << to_string(p) << " Est: " << to_string(est_pm25) << "\n";
			}
		}				
	}

	cout << "\nFreeing host memory...\n";
	free(trainingData);
	free(testData);	
	free(modData);
	free(pm25data);	
	
	cout << "Freeing device memory...\n";
	hipFree(d_trainingData);
	hipFree(d_pm25data);

	hipDeviceReset();

	return 0;
}

void loadData() {
	fstream file;
	string line;
	int i = 0;

	file.open("pm25_2009_data.csv");	
	while (getline(file, line, '\n') && i < dataSize) {
		istringstream templine(line);
		string data;		

		getline(templine, data, ',');
		pm25data[i].t = atoi(data.c_str());

		getline(templine, data, ',');
		pm25data[i].y = atof(data.c_str());

		getline(templine, data, ',');
		pm25data[i].x = atof(data.c_str());

		getline(templine, data, ',');
		pm25data[i].pm25 = atof(data.c_str());

		pm25data[i].d = 0;		

		i++;
	}
	file.close();
}

void seqDistanceCalc(SITE *data, int dataSize, double x, double y, int t, double c) {
	double dt, dy, dx;

	for (int i = 0; i < dataSize; i++) {
		dt = data[i].t - t;
		dx = data[i].x - x;
		dy = data[i].y - y;

		data[i].d = (dx*dx) + (dy*dy) + (dt*dt)*(c*c);

		// d > 1.4, t > 7
		if (data[i].d > 1.96 || dt > 49)
			data[i].d = INFINITY;

		//if (data[i].d != modData[i].d && data[i].d != INFINITY)
		//	cout << "Result " << to_string(i) << " is not the same!\n";
	}
}

double idwCalc(int N, double p, SITE *data) {
	double w, wi;
	double lambdaD, lambda;

	w = 0;
	for (int i = 0; i < N; i++) {
		wi = data[i].pm25;

		lambdaD = 0;
		for (int k = 0; k < N; k++)
			lambdaD += pow(1 / sqrt(data[k].d), p);

		lambda = pow(1 / sqrt(data[i].d), p) / lambdaD;

		w += wi*lambda;
	}
	return w;
}